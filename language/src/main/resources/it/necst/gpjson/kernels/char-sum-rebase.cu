
#include <hip/hip_runtime.h>
__global__ void f(char *charArr, int n, char *base, int offset, char *charNewArr) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elemsPerThread = (n+stride-1) / stride;

    long start = index * elemsPerThread;
    long end = start + elemsPerThread;

    for (long i = start; i < end && i < n; i++) {
        charNewArr[i+offset] = charArr[i] + base[index];
    }
}